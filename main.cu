#include "hip/hip_runtime.h"
#include<vector>
#include<map>
#include<algorithm>
#include<iostream>
#include<fstream>
#include<utility>
#include<vector>
#include<list>

#include"vec_utils.h"
#include"cudaerror.h"
#include"data.h"
#include"parameters.h"
#include"listUpdaters.cuh"


Parameters params;
std::list<ListUpdater*> listUpdaters;

int main(int argc, char** argv)
{
    d_Data d_part;
    params.device = 0;
    params.block_size = 128;
    hipSetDevice(params.device);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 900000000);

    d_part.atom_count = 100;
    d_part.atom_count = d_part.atom_count;
    int *t = (int*)calloc(d_part.atom_count, sizeof(int));
    float3 *c = (float3*)calloc(d_part.atom_count, sizeof(float3));
    for(int i = 0; i < d_part.atom_count; i++)
    {
        t[i] = TYPEA;
        c[i].x = i;
    }
    hipMalloc((void**)&(d_part.t), d_part.atom_count * sizeof(int));
    hipMalloc((void**)&(d_part.c), d_part.atom_count * sizeof(float3));
    hipMemcpy(d_part.t, t, d_part.atom_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_part.c, c, d_part.atom_count * sizeof(float3), hipMemcpyHostToDevice);
    d_part.atom_count = d_part.atom_count;

    params.maxpossiblepairs = 1000;
    hipMalloc((void**)&(d_part.possiblePairsCount), d_part.atom_count * sizeof(int));
    hipMemset(d_part.possiblePairsCount, 0, d_part.atom_count * sizeof(int));
    hipMalloc((void**)&(d_part.possiblePairs), d_part.atom_count * params.maxpossiblepairs * sizeof(int));
    hipMemset(d_part.possiblePairs, 0, d_part.atom_count * params.maxpossiblepairs * sizeof(int));
    checkCUDAError(__LINE__,__FILE__);
    hipDeviceSynchronize();
    params.possiblepairscutoff = 2;
    listUpdaters.push_back(new ListUpdater(d_part, params));
    listUpdaters.front()->update();
}
