#include "hip/hip_runtime.h"
#include<stdio.h>

#include"data.h"


__global__ void possibleListUpdater_kernel(d_Data data, IndexStruct selected)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if( (i < data.atom_count) )
    {
        if(i < selected.index_count)
            i = selected.index[i];
        for(int j = 0; j < data.atom_count; j++)
        {
            if ( (j != i) && (data.t[j] != TYPENONE) )
            {
                int3 rj = data.c[j];
                data.possiblePairsCount[i] = rj.x;
            }
        }
    }
}

void checkCUDAError(int line, const char *file)
{
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
    {
        if(line >= 0)
    		printf("CUDA error: %s  at line %d file %s\n", hipGetErrorString(error), line, file);
        else
    		printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(0);
	}
}

IndexStruct selected;

int main(int argc, char** argv)
{
    d_Data d_part;
    d_part.block_size = 128;
    hipSetDevice(3);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 900000000);

    d_part.atom_count = 100;
    d_part.atom_count = d_part.atom_count;
    int *t = (int*)calloc(d_part.atom_count, sizeof(int));
    int3 *c = (int3*)calloc(d_part.atom_count, sizeof(int3));
    for(int i = 0; i < d_part.atom_count; i++)
    {
        t[i] = TYPEA;
        c[i].x = (float)i;
    }
    hipMalloc((void**)&(d_part.t), d_part.atom_count * sizeof(int));
    hipMalloc((void**)&(d_part.c), d_part.atom_count * sizeof(int3));
    hipMemcpy(d_part.t, t, d_part.atom_count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_part.c, c, d_part.atom_count * sizeof(int3), hipMemcpyHostToDevice);
    d_part.atom_count = d_part.atom_count;

    hipMalloc((void**)&(d_part.possiblePairsCount), d_part.atom_count * sizeof(int));
    hipMemset(d_part.possiblePairsCount, 0, d_part.atom_count * sizeof(int));
    checkCUDAError(__LINE__,__FILE__);
    hipDeviceSynchronize();
    selected.index_count = -1;
    selected.index = NULL;
    int grid_size;
    grid_size = d_part.atom_count/d_part.block_size+1;
    possibleListUpdater_kernel<<<grid_size, d_part.block_size>>>(d_part, selected);
    hipDeviceSynchronize();
    checkCUDAError(__LINE__,__FILE__);
}
