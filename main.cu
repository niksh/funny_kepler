#include<vector>
#include<map>
#include<algorithm>
#include<iostream>
#include<fstream>
#include<utility>
#include<vector>
#include<list>

#include"vec_utils.h"
#include"cudaerror.h"
#include"data.h"
#include"parameters.h"
#include"listUpdaters.cuh"


Parameters params;
std::list<ListUpdater*> listUpdaters;

int main(int argc, char** argv)
{
    Data data;
    d_Data d_part;
    params.device = 0;
    params.block_size = 128;
    hipSetDevice(params.device);
    hipDeviceSetLimit(hipLimitPrintfFifoSize, 900000000);

    params.max_parts = 100;
    data.atom_count = params.max_parts;
    data.c.resize(data.atom_count);
    data.t.resize(data.atom_count);
    for(int i = 0; i < data.atom_count; i++)
    {
        data.t.h[i] = TYPEA;
        data.c.h[i].x = i;
    }
    data.c.h2d();
    data.t.h2d();
    d_part.c = data.c.d_ptr();
    d_part.t = data.t.d_ptr();
    d_part.atom_count = data.atom_count;

    params.maxpossiblepairs = 1000;
    hipMalloc((void**)&(d_part.possiblePairsCount), data.atom_count * sizeof(int));
    hipMemset(d_part.possiblePairsCount, 0, data.atom_count * sizeof(int));
    hipMalloc((void**)&(d_part.possiblePairs), data.atom_count * params.maxpossiblepairs * sizeof(int));
    hipMemset(d_part.possiblePairs, 0, data.atom_count * params.maxpossiblepairs * sizeof(int));
    checkCUDAError(__LINE__,__FILE__);
    hipDeviceSynchronize();
    data.d_data = d_part;
    params.possiblepairscutoff = 2;
    listUpdaters.push_back(new ListUpdater(&data, &params));
    listUpdaters.front()->update();
}
