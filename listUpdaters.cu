#include "hip/hip_runtime.h"
#include"listUpdaters.cuh"
#include"vec_utils.h"
#include"cudaerror.h"

__global__ void possibleListUpdater_kernel(d_Data data, Parameters params, IndexStruct select)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if( (i < data.atom_count) && (data.t[i] != TYPENONE) )
    {
        if(i < select.index_count)
            i = select.index[i];
        float3 ri = data.c[i];
        int pairsCount = 0;
        for(int j = 0; j < data.atom_count; j++)
        {
            if ( (j != i) && (data.t[j] != TYPENONE) )
            {
                float3 rj = data.c[j];
                if( (len(ri-rj) < params.possiblepairscutoff) && (pairsCount < params.maxpossiblepairs) )
                    pairsCount ++;
            }
        }
        data.possiblePairsCount[i] = pairsCount;
        printf("%d %d\n", i, pairsCount);
    }
}

void ListUpdater::update(IndexStruct select)
{
    int grid_size;
    if(select.index_count > 0)
        grid_size = select.index_count/params->block_size+1;
    else
        grid_size = data->atom_count/params->block_size+1;
    possibleListUpdater_kernel<<<grid_size, params->block_size>>>(data->d_data, *params, select);
    hipDeviceSynchronize();
    checkCUDAError(__LINE__,__FILE__);
}
