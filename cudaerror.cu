#include<stdio.h>
#include"cudaerror.h"
void checkCUDAError(int line, const char *file)
{
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
    {
        if(line >= 0)
    		printf("CUDA error: %s  at line %d file %s\n", hipGetErrorString(error), line, file);
        else
    		printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(0);
	}
}
